#include "hip/hip_runtime.h"
%%writefile cuda_example2.cu
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

#define SIZE 1010228

__device__ void strandSort(int* ip, int* op, int size) {
    int tid = threadIdx.x;
    int step = blockDim.x;

    for (int i = tid; i < size; i += step) {
        int current = ip[i];
        int j = i - 1;

        // Move elements greater than current to the right
        while (j >= 0 && op[j] > current) {
            op[j + 1] = op[j];
            j--;
        }

        // Insert the current element in the correct position
        op[j + 1] = current;
    }
}

__global__ void kernelStrandSort(int* ip, int* op, int size) {
    extern __shared__ int sublist[];

    int tid = threadIdx.x;

    for (int i = 0; i < size; ++i) {
        sublist[tid] = ip[i];
        __syncthreads();  // Synchronize threads after updating sublist

        strandSort(sublist, op, i + 1);

        // Wait for all threads to finish sorting before updating sublist
        __syncthreads();
    }
}

int main() {
    // Read input from a file
    std::ifstream inputFile("/content/drive/MyDrive/T10I4D100K.dat.txt");
    if (!inputFile.is_open()) {
        std::cerr << "Error opening the input file." << std::endl;
        return 1;
    }

    std::vector<int> inputVector;
    int inputValue;
    while (inputFile >> inputValue) {
        inputVector.push_back(inputValue);
    }
    if (inputVector.size() != SIZE) {
        std::cerr << "Input size does not match the defined size (" << SIZE << ")." << std::endl;
        return 1;
    }

    int* ip;
    hipMallocManaged(&ip, SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        ip[i] = inputVector[i];
    }

    int* op;
    hipMallocManaged(&op, SIZE * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int blockSize = 256;
    int gridSize = (SIZE + blockSize - 1) / blockSize;

    kernelStrandSort<<<blockSize, gridSize, SIZE * sizeof(int)>>>(ip, op, SIZE);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float seconds = 0;
    hipEventElapsedTime(&seconds, start, stop);


    std::cout << "Time taken by the kernel: " << seconds << " s" << std::endl;

    hipFree(ip);
    hipFree(op);

    return 0;
}

